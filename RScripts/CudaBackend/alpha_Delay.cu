#include "hip/hip_runtime.h"
#include <R.h>
// #include <iostream>
#include <cstdlib>
#include <ctime>
#include <iostream>

#define HI 0.35
#define LO 0.25

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
/**
 * Cuda kernel to calculate delay based on alpha power law.
 * Assumption : vdd (x) / Vth(y)
 * ------------------------------
 *    | vth1  | vth2  | vth3  |
 * ------------------------------
 * vdd1 | D1  | D2  | D3  |
 * ------------------------------
 * vdd2 | D4  | D5  | D6  |
 * ------------------------------
 *
 * Delay = (vdd/(vdd-vth)^alpha)
 * @param vdd      gpu vdd array
 * @param vth      gpu vth array
 * @param alpha    gpu alpha factor
 * @param Constant gpu constant to scale delay
 * @param _DELAY_  return pointer
 * @param vdd_l    length of vdd
 * @param vth_l    length of vth
 */
extern "C" __global__ void delay(const double *vdd, const double *vth, const double alpha, const double Constant, double *_DELAY_,
                      const int vdd_l, const int vth_l) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int bDim = blockDim.x;
  int index = tid + (bid * bDim);
  int vddID, vthID;
  vddID = (index / vth_l);
  vthID = index % vth_l;
  double localvdd, localvth;
  if (index < (vdd_l * vth_l)) {
    localvdd = vdd[vddID];
    localvth = vth[vthID];
    _DELAY_[index] = 1/((Constant * localvdd) / pow((localvdd - localvth), alpha));
  }
}

/////////////////////////////////////
// Define interface function for R //
/////////////////////////////////////
// extern "C" 
// void alpha_power_law(double *vdd, double *vth, double *alpha, double *Constant, double *_DELAY_, int *vddl, int *vthl);

extern "C" 
void alpha_power_law(double *vdd, double *vth, double *alpha, double *Constant, double *_DELAY_, int *vddl, int *vthl) {
  ///////////////////
  // Device memory //
  ///////////////////
  double *d_vdd, *d_vth, *d_DELAY_;
  int vdd_l, vth_l;
  vdd_l = (*vddl);
  vth_l = (*vthl);

  int totalDelays = vdd_l * vth_l;
//  std::cout << "totalDelays=" << totalDelays << std::endl;
//  std::cout << "vddl=" << vdd_l << std::endl;
//  std::cout << "vthl=" << vth_l << std::endl;
//  std::cout << "Constant=" << *Constant << std::endl;
//  std::cout << "alpha=" << *alpha << std::endl;
  //////////////////////////
  // Define configuration //
  //////////////////////////
  int blockSize; // The launch configurator returned block size
  // int minGridSize; // The minimum grid size needed to achieve the
  // maximum occupancy for a full device launch
  int gridSize; // The actual grid size needed, based on input size
  
  blockSize = 1024;
  //hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, delay, 0, totalDelays);
  
  // Round up according to array size
  gridSize = (totalDelays + blockSize - 1) / blockSize;
//  std::cout << "grid x block " << gridSize << "x" << blockSize << std::endl;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  gpuErrchk(hipMalloc((void **)&d_vdd, vdd_l * sizeof(double)));
  gpuErrchk(hipMalloc((void **)&d_vth, vth_l * sizeof(double)));
  gpuErrchk(hipMalloc((void **)&d_DELAY_, totalDelays * sizeof(double)));

  gpuErrchk(hipMemcpy(d_vdd, vdd, vdd_l * sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_vth, vth, vth_l * sizeof(double), hipMemcpyHostToDevice));
  // hipMemcpy(d_DELAY_, _DELAY_, totalDelays * sizeof(double), hipMemcpyHostToDevice);
  
  hipEventRecord(start);
  delay<<<gridSize, blockSize>>> (d_vdd, d_vth, (*alpha), (*Constant), d_DELAY_, vdd_l, vth_l);
  gpuErrchk( hipPeekAtLastError() );
  // hipDeviceSynchronize();
  hipMemcpy(_DELAY_, d_DELAY_, totalDelays * sizeof(double), hipMemcpyDeviceToHost);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float ms = 0.0;
  hipEventElapsedTime(&ms, start, stop);

  //////////////////
  // Debug prints //
  //////////////////
  /*
  for (int i = 0; i < vdd_l; i++){
    std::cout << vdd[i] <<"|";
  }
  std::cout << std::endl;
  for(int j= 0; j < vth_l; j++){
    std::cout << vth[j] << "|";
  }
  std::cout << std::endl;
*/
 /* int index = 0;
  for (int i = 0; i < vdd_l; i++){
    for(int j= 0; j < vth_l; j++)
    {
      index =(i * vth_l) + j;
      std::cout << _DELAY_[index] << "|";
    }
    std::cout << "\n";
  }*/

  
  std::cout << "Elapsed " << ms << "ms" << std::endl;

  hipFree(d_vdd);
  hipFree(d_vth);
  hipFree(d_DELAY_);
  // hipDeviceReset();
}


#ifdef NRTEST
int main() {
  int vddl = 501;
  int vthl = 90000;

  double vddArray[vddl];
  double vddStep = (1.0 - 0.5) / vddl;
  double vddi = 0.5;

  for (int i = 0; i < vddl; i++) {
    vddArray[i] = vddi;
    vddi += vddStep;
  }

  double vthArray[vthl];
  double vthStep = (HI - LO) / vthl;
  double vthi = LO;

  for (int i = 0; i < vthl; i++) {
    vthArray[i] = vthi;
    vthi += vthStep;
  }

  double alpha = 1.1;
  double constant = 8e-10;
  int totalCount = vthl * vddl;
  std::cout << "totalCount = " << totalCount << "\n";
  double *result =new double[totalCount];
  for (int i = 0; i < totalCount; i++)
  {
    // std::cout << "|i" << i;
    result[i] = 1.0;
  }

  std::cout << std::endl;
  // double vddl_f, vthl_f;
  // vddl_f = (double)vddl;
  // vthl_f = (double)vthl;
  // alpha_power_law(vddArray, vthArray,  &alpha, &constant,result, &vddl_f, &vthl_f);
  alpha_power_law(vddArray, vthArray,  &alpha, &constant,result, &vddl, &vthl);
/*
  int index = 0;
  for (int i = 0; i < vddl; i++){
    for(int j= 0; j < vthl; j++)
    {
      index =(i * vthl) + j;
      std::cout << result[index] << "|";
    }
    std::cout << "\n";
  }
*/
  delete[] result;
  return 0;
}
#endif
